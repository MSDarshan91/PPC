#include "hip/hip_runtime.h"
#include "cp.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <unistd.h>
using namespace std;
using namespace std::chrono;
#define BLOCK_SIZE 16

#define CHECK_CUDA_ERROR(call) do { \
        hipError_t result_ = (call); \
        if (result_ != hipSuccess) { \
            fprintf(stderr, #call " failed: %s\n", \
                    hipGetErrorString(result_)); \
            exit(1); \
        } \
    } while(0)
__global__ void var(float *input,float *output, int N, float mean)
	{
 
      int idx=threadIdx.x+(blockDim.x*blockIdx.x);
      if (idx < N) output[idx] = (input[idx]-mean)*(input[idx]-mean);
    }
__global__ void norm(float *input, int N,float mean,float sd)
	{
 
      int idx=threadIdx.x+(blockDim.x*blockIdx.x);
      if (idx < N) input[idx] =  (input[idx]-mean)/sd;
    }


__global__ void matrixMultiply(float * A, float * C, int ny, int nx) 
{
	if(blockIdx.x < blockIdx.y)
		return;
  __shared__ float ds_M[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float ds_N[BLOCK_SIZE][BLOCK_SIZE];
  int bx= blockIdx.x; 
  int by= blockIdx.y;
  int tx= threadIdx.x; 
  int ty= threadIdx.y;
  int Row= by * BLOCK_SIZE + ty; 
  int Col= bx * BLOCK_SIZE + tx; 
  float Pvalue= 0;
  int n_tiles = nx/BLOCK_SIZE + (nx%BLOCK_SIZE == 0 ? 0:1);
  for (int m= 0; m < n_tiles; ++m) 
  {
    if (Row < ny && m*BLOCK_SIZE+tx < nx)
      ds_M[ty][tx] = A[Row*nx + m*BLOCK_SIZE+tx];
    else
      ds_M[ty][tx] = 0;
 
    if (Col < ny && m*BLOCK_SIZE+ty < nx)
      ds_N[ty][tx] = A[Col*nx + m*BLOCK_SIZE+ty];
    else
      ds_N[ty][tx] = 0;
 
    __syncthreads();
	if(Col<Row)
      continue;
    for (int k = 0; k < BLOCK_SIZE; ++k)
      Pvalue += ds_M[ty][k] * ds_N[k][tx];
    __syncthreads();
  }
  if (Row < ny && Col < ny)
    C[Row*ny+Col] = Pvalue;
}
 
void correlate(int ny, int nx, const float* data, float* result) 
{
	float* inter = new float[ny*nx];
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	for (int y = 0; y < ny; ++y) 
	{
		float *a_d_input;
		float *a_h;
		float *a_d_output;	
		float mean = 0.0;
		float sd = 0.0;
		size_t size = nx * sizeof(float);
		//Finding the mean
		for (int x = 0; x < nx; ++x) 
		{
			inter[x + y*nx] = data[x + y*nx];
			mean += inter[x + y*nx];
			//cout<<data[x+y*nx];
		}
		mean = mean/nx;
		//cout<<"Mean=="<<" "<<mean<<endl;
		//Finding the Standard Deviation
		a_h =  new float[nx];
		CHECK_CUDA_ERROR(hipMalloc((void **) &a_d_input, size)); 
		CHECK_CUDA_ERROR(hipMalloc((void **) &a_d_output, size)); 
		CHECK_CUDA_ERROR(hipMemcpy(a_d_input, &inter[y*nx], size, hipMemcpyHostToDevice));
		int block_size = 400;
		int n_blocks = nx/block_size + (nx%block_size == 0 ? 0:1);
		var<<< n_blocks, block_size >>> (a_d_input,a_d_output,nx,mean);
		CHECK_CUDA_ERROR(hipGetLastError());
		CHECK_CUDA_ERROR(hipMemcpy(a_h, a_d_output, size, hipMemcpyDeviceToHost));
		//cout<<"Inter"<<endl;
		for (int x = 0; x < nx; ++x) 
		{
			sd += a_h[x];
		}
		//cout<<endl;
		sd= sqrt(sd);
		
		//hipMalloc((void **) &a_d, size); 
		CHECK_CUDA_ERROR(hipMemcpy(a_d_output, &inter[y*nx], size, hipMemcpyHostToDevice));
		norm<<< n_blocks, block_size >>> (a_d_output,nx,mean,sd);
		CHECK_CUDA_ERROR(hipGetLastError());
		CHECK_CUDA_ERROR(hipMemcpy(&inter[y*nx], a_d_output, size, hipMemcpyDeviceToHost));
		//cout<<"SD=="<<" "<<sd<<endl;
		//Finding zero mean and unit variance
		
   	}
	high_resolution_clock::time_point t2 = high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds>( t2 - t1 ).count();
	//cout << duration<<endl;
//cout<<"Done"<<endl;
	float *d_A;
	float *d_C;
	size_t size = nx*ny*sizeof(float);
	//float* inter_t = new float[ny*nx];
	//cout<<"Inter"<<endl;
	//for(int i =0;i< nx*ny;i++)
		//cout<< inter[i]<<" ";
	//cout<<endl;
	CHECK_CUDA_ERROR(hipMalloc((void**) &d_A, size));
//	CHECK_CUDA_ERROR(hipMalloc((void**) &d_B, size));
	CHECK_CUDA_ERROR(hipMalloc((void**) &d_C, ny*ny*sizeof(float)));
	CHECK_CUDA_ERROR(hipMemcpy(d_A, inter, size,hipMemcpyHostToDevice));
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
	int n_blocks = ny/BLOCK_SIZE + (ny%BLOCK_SIZE == 0 ? 0:1);
   	dim3 grid(n_blocks,n_blocks);
	high_resolution_clock::time_point t3 = high_resolution_clock::now();
	duration = std::chrono::duration_cast<std::chrono::microseconds>( t3 - t2 ).count();
	
	matrixMultiply<<< grid, threads >>>(d_A, d_C,ny, nx);
	CHECK_CUDA_ERROR(hipGetLastError());
	CHECK_CUDA_ERROR(hipMemcpy(result, d_C, ny*ny*sizeof(float), hipMemcpyDeviceToHost));
	high_resolution_clock::time_point t4 = high_resolution_clock::now();
	duration = std::chrono::duration_cast<std::chrono::microseconds>( t4 - t3 ).count();
	cout << duration<<endl;
	//cout<<endl;	
	//for(int i =0;i< ny*ny;i++)
	//	cout<< result[i]<<" ";	
	//cout<<endl;
}
