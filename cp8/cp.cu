#include "hip/hip_runtime.h"
#include "cp.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
#define BLOCK_SIZE 8
#define CHECK_CUDA_ERROR(call) do { \
        hipError_t result_ = (call); \
        if (result_ != hipSuccess) { \
            fprintf(stderr, #call " failed: %s\n", \
                    hipGetErrorString(result_)); \
            exit(1); \
        } \
    } while(0)
__global__ void normalize(float *input,float *output, int ny, int nx)
	{
		float mean = 0.0;
		float sd = 0.0;
		int tx = threadIdx.x + (blockDim.x * blockIdx.x);
		int ty = threadIdx.y + (blockDim.y * blockIdx.y);
		int row = ty*nx;
		if(tx>= nx || ty>=ny)
			return;
		output[ty * nx + tx] = input[ty * nx + tx];
		for(int i=0;i<nx;i++)
			mean+=input[row+i];
		mean = mean/nx;
		float temp=0.0;
		for(int i=0;i<nx;i++)
			temp+=((input[row+i]-mean)*(input[row+i]-mean));
		sd = sqrt(temp);
		output[ty * nx + tx] = (output[ty * nx + tx] - mean)/sd;
    }

__global__ void matrixMul( float* C, float* A, int ny,int nx)
{
   int tx = threadIdx.x + (blockDim.x * blockIdx.x);
   int ty = threadIdx.y + (blockDim.y * blockIdx.y);
  
   if(tx>= ny || ty>=ny)
    return;
   float value = 0;
   for (int i = 0; i < nx; ++i)
   {
      float elementA = A[ty * nx + i];
      float elementB = A[tx * nx + i];
      value += elementA * elementB;
   }
    C[ty * ny + tx] = value;
}
 
void correlate(int ny, int nx, const float* data, float* result) 
{
	float* inter = new float[ny*nx];
	float *a_d_input;
	float *a_d_output;
	CHECK_CUDA_ERROR(hipMalloc((void **) &a_d_input, nx*ny*sizeof(float))); 
	CHECK_CUDA_ERROR(hipMalloc((void **) &a_d_output, nx*ny*sizeof(float))); 
	CHECK_CUDA_ERROR(hipMemcpy(a_d_input, data, nx*ny*sizeof(float), hipMemcpyHostToDevice));
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
	int nx_blocks = nx/BLOCK_SIZE + (nx%BLOCK_SIZE == 0 ? 0:1);
	int ny_blocks = ny/BLOCK_SIZE + (ny%BLOCK_SIZE == 0 ? 0:1);
   	dim3 grid(nx_blocks,ny_blocks);
	normalize<<< grid, threads >>>(a_d_input, a_d_output, ny,nx);
	CHECK_CUDA_ERROR(hipGetLastError());
	CHECK_CUDA_ERROR(hipMemcpy(inter, a_d_output, nx*ny*sizeof(float), hipMemcpyDeviceToHost));
	float *d_A;
	float *d_C;
	size_t size = nx*ny*sizeof(float);
//	float* inter_t = new float[ny*nx];
	//cout<<"Inter"<<endl;
	//for(int i =0;i< nx*ny;i++)
		//cout<< inter[i]<<" ";
	//cout<<endl;
	CHECK_CUDA_ERROR(hipMalloc((void**) &d_A, size));
//	CHECK_CUDA_ERROR(hipMalloc((void**) &d_B, size));
	CHECK_CUDA_ERROR(hipMalloc((void**) &d_C, ny*ny*sizeof(float)));
	CHECK_CUDA_ERROR(hipMemcpy(d_A, inter, size,hipMemcpyHostToDevice));
	//cout<<n_blocks<<" "<< nx<<endl;
   	dim3 grid_1(ny_blocks,ny_blocks);
 	matrixMul<<< grid_1, threads >>>(d_C, d_A, ny,nx);
	CHECK_CUDA_ERROR(hipGetLastError());
	CHECK_CUDA_ERROR(hipMemcpy(result, d_C, ny*ny*sizeof(float), hipMemcpyDeviceToHost));
	//cout<<endl;	
	//for(int i =0;i< ny*ny;i++)
	//	cout<< result[i]<<" ";	
	//cout<<endl;
}
